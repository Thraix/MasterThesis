#include "hip/hip_runtime.h"
#pragma once

#include "Common.cu"

#include "VectorMath.cu"

#if __YCM__
#define _BRESENHAM
#endif

namespace ILV
{

#if defined(_ILV)
#include "VoxelizeILV.cu"
#elif defined(_BRESENHAM)
#include "VoxelizeBresenham.cu"
#else
#error No Algorithm specified
#endif

  __device__
  int3 getNextScanline(LineData& data, int3 pos, int2 diff, int axis, int axisX, int axisY, bool increaseScanline)
  {
    int z = get(pos, axis);
    int3 nextPos = data.pos;

    int2 increment = make_int2(diff.y, -diff.x);
    int c = get(pos, axisX) * increment.x + get(pos, axisY) * increment.y;
    int cLower = c - abs(diff.x) - abs(diff.y);
    int cUpper = c + abs(diff.x) + abs(diff.y);
    int cEnd = get(data.end, axisX) * increment.x + get(data.end, axisY) * increment.y;
    int cNext = get(nextPos, axisX) * increment.x + get(nextPos, axisY) * increment.y;

    // This resolves holes in the voxelization in some edge cases when
    // scanline doesn't fill all the voxels. This happens when a new slice
    // is started and the scanline goes backwards.
    // This then looks for that problem and limits the scanline to only
    // be behind the old one
    if((cEnd - c) * (cNext - c) < 0)
    {
      if(cEnd > c)
        cUpper = c;
      else
        cLower = c;
    }
    else if(!increaseScanline)
    {
      return pos;
    }

    while(get(nextPos, axis) == z && pos != data.end && cLower <= c && c <= cUpper)
    {
      pos = nextPos;
      if(data.pos != data.end)
        nextPos = GetNextVoxelInLine(data);
      c = get(nextPos, axisX) * increment.x + get(nextPos, axisY) * increment.y;
    }
    return pos;
  }

  // Check if the 'inside' line is between 'vec1' and 'vec2'.
  // Source: https://stackoverflow.com/a/17497339
  __device__
  bool VectorInside(int2 vec1, int2 vec2, int2 inside)
  {
    return
      ((vec1.y * inside.x - vec1.x * inside.y) * (vec1.y * vec2.x - vec1.x * vec2.y) >= 0 && (vec2.y * inside.x - vec2.x * inside.y) * (vec2.y * vec1.x - vec2.x * vec1.y) >= 0) ||
      ((-vec1.y * inside.x + vec1.x * inside.y) * (vec1.y * vec2.x - vec1.x * vec2.y) >= 0 && (-vec2.y * inside.x + vec2.x * inside.y) * (vec2.y * vec1.x - vec2.x * vec1.y) >= 0);
  }

  __device__
  void voxelizeInterior(int3 v1, int3 v2, int3 v3, int axis)
  {
    int axisX = (axis + 1) % 3;
    int axisY = (axis + 2) % 3;

    int2 scanlineDir = toInt2(cross(v2 - v1, v3 - v1), axisY, axisX);
    scanlineDir.x = -scanlineDir.x;

    bool swapped = false;
    LineData dataFrom = CreateLineData(v1, v2);
    LineData dataTo = CreateLineData(v1, v3);
    LineData dataEdge = CreateLineData(v2, v3);

    int3 posFrom = dataFrom.pos;
    int3 posTo = dataTo.pos;

    int2 diff = toInt2(posTo - posFrom, axisX, axisY);

    int2 d13 = toInt2(v3 - v1, axisX, axisY);
    int2 d12 = toInt2(v2 - v1, axisX, axisY);

    int increaseScanline = true;
    if(get(v1, axis) != get(v3, axis))
      diff = scanlineDir;
    else
    {
      swapped = true;
      dataFrom = dataEdge;
      posFrom = dataFrom.pos;
      diff = toInt2(posTo - posFrom, axisX, axisY);
      increaseScanline = false;
    }

    while(posTo != v3 || posFrom != v3)
    {
      posFrom = getNextScanline(dataFrom, posFrom, diff, axis, axisX, axisY, increaseScanline);
      posTo = getNextScanline(dataTo, posTo, diff, axis, axisX, axisY, increaseScanline);
      increaseScanline = true;

      voxelizeLine(posFrom, posTo, 1);

      if(posFrom == v2 && !swapped)
      {
        dataFrom = dataEdge;
        swapped = true;
      }

      diff = toInt2(posTo - posFrom, axisX, axisY);
      if(get(posFrom, axis) != get(dataFrom.pos, axis) && get(posTo, axis) != get(dataTo.pos, axis))
      {
        posFrom = dataFrom.pos;
        posTo = dataTo.pos;

        if(dataFrom.pos != dataFrom.end)
          GetNextVoxelInLine(dataFrom);
        if(dataTo.pos != dataTo.end)
          GetNextVoxelInLine(dataTo);

        increaseScanline = false;
      }
      if(!swapped && VectorInside(d13, d12, diff))
      {
        diff = scanlineDir;
      }
    }
  }

  __device__
  void voxelize(float3 v1, float3 v2, float3 v3)
  {
    int3 vi1 = toInt3(v1);
    int3 vi2 = toInt3(v2);
    int3 vi3 = toInt3(v3);

    // Calculate most dominant axis
    int3 normal = abs(cross(vi2 - vi1, vi3 - vi1));
    int domNormal = max(normal);
    int axis = 0;
    if(domNormal == normal.x) axis = 0;
    else if(domNormal == normal.y) axis = 1;
    else if(domNormal == normal.z) axis = 2;

    // Sort by most dominant axis
    if(get(vi1, axis) > get(vi2, axis))
      swap(vi1,vi2);
    if(get(vi1, axis) > get(vi3, axis))
      swap(vi1,vi3);
    if(get(vi2, axis) > get(vi3, axis))
      swap(vi2,vi3);

    // Voxelize edges
    voxelizeLine(vi1, vi2, 1);
    voxelizeLine(vi2, vi3, 1);
    voxelizeLine(vi1, vi3, 1);

    // Voxelize interior
    voxelizeInterior(vi1, vi2, vi3, axis);
  }
}
