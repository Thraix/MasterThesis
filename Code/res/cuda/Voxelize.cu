#include "hip/hip_runtime.h"
#include "Common.cu"

#if defined(_RLV)
#include "VoxelizeRLV.cu"
#else
#include "VoxelizeInteger.cu"
#endif

extern "C" __global__
void voxelize(int* indices, int indexCount, float3* vertices, int vertexCount, bool first)
{
  compareFirst = first;
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  if(index * 3 + 2 > indexCount)
    return;
  int i1 = indices[index * 3];
  int i2 = indices[index * 3 + 1];
  int i3 = indices[index * 3 + 2];

  float3 v1 = modelMatrix * vertices[i1];
  float3 v2 = modelMatrix * vertices[i2];
  float3 v3 = modelMatrix * vertices[i3];

#if defined(_RLV)
    RLV::voxelize(v1, v2, v3);
#elif defined(_ILV)
    ILV::voxelize(v1, v2, v3);
#elif defined(_BRESENHAM)
    ILV::voxelize(v1, v2, v3);
#else
#error No Algorithm specified
#endif
}

