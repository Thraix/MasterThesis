#include "hip/hip_runtime.h"
surface<void, 3> voxelGrid;
__constant__ int voxelGridSize;

extern "C" __global__
void compare(int32_t* result)
{
  // This is not how you should code a GPU....
  // But downloading GB of data to the CPU RAM
  // gave me segfaults...
  int offsetDim = ceil(voxelGridSize / (float)blockDim.x);
  int index = threadIdx.x * offsetDim;
  int intersection = 0;
  int inFirst = 0;
  int inSecond = 0;
  for(int z = index;z<index+offsetDim && z<voxelGridSize;z++)
  {
    for(int y = 0;y<voxelGridSize;y++)
    {
      for(int x = 0;x<voxelGridSize;x++)
      {
        unsigned char color;
        surf3Dread(&color, voxelGrid, x, y, z);
        if(color == 3)
          intersection++;
        else if(color == 2)
          inFirst++;
        else if(color == 4)
          inSecond++;
        else if(color != 0)
          printf("Unmatched %d!\n", color);
      }
    }
  }
  atomicAdd(&result[0], intersection);
  atomicAdd(&result[1], inFirst);
  atomicAdd(&result[2], inSecond);
}
