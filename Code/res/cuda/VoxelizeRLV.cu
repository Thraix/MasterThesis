#include "hip/hip_runtime.h"
#pragma once

#include "Common.cu"

#include "VectorMath.cu"

namespace RLV
{
  struct ScanlineCache
  {
    float3 v1;
    float3 v2;
    float3 v3;

    float3 unitDir12;
    float3 unitDir13;
    float3 unitDir23;


    float2 projUnitDir12;
    float2 projUnitDir13;
    float2 projUnitDir23;

    float lengthDir12;
    float lengthDir13;
    float minLengthDir23;
    float lengthDir23;

    float scanlineLength;
    float scanlineInvDirAxis;
    float scanlineMaxLength;

    float slInvDotDir12; //  1 / dot(projSlDir, projUnitDir12)
    float slInvDotDir13; //  1 / dot(projSlDir, projUnitDir13)
    float slInvDotDir23; //  1 / dot(projSlDir, projUnitDir23)
  };

  __device__
  void voxelizeLine(float3 v1, float3 v2, unsigned char color)
  {
    float3 dir = v2 - v1;
    float dirLength = length(dir);
    float3 unitDir = normalize(dir);
    float3 signDir = sign(dir);

    float3 nextPlane = make_float3(
        dir.x < 0 ? ceil(v1.x - 1) : floor(v1.x + 1),
        dir.y < 0 ? ceil(v1.y - 1) : floor(v1.y + 1),
        dir.z < 0 ? ceil(v1.z - 1) : floor(v1.z + 1)
        );

    int3 pos = make_int3(
        dir.x < 0 ? nextPlane.x : nextPlane.x-1,
        dir.y < 0 ? nextPlane.y : nextPlane.y-1,
        dir.z < 0 ? nextPlane.z : nextPlane.z-1
        );
    voxelizePoint(pos, color);


    float3 t = (nextPlane - v1) / unitDir;
    float3 tStep = 1.0f / abs(unitDir);
    float maxStep = abs(dir.x) + abs(dir.y) + abs(dir.z) + 3.0f;

    int i = 0;
    float stepLength = 0;

    while(pos != toInt3(v2) && stepLength < dirLength)
    {
      float tmin = min(t);
      int axis = 0;
      if(tmin == t.x) axis = 0;
      else if(tmin == t.y) axis = 1;
      else if(tmin == t.z) axis = 2;

      t -= tmin;
      stepLength += tmin;

      set(pos, axis, get(pos, axis) + get(signDir, axis));
      voxelizePoint(pos, color);
      set(t, axis, get(tStep, axis));
      i++;
    }
  }

  __device__ 
  void calculateScanline(float3 v1, float3 v2, float3 v3, int axis, ScanlineCache& cache)
  {
    float3 dir12 = v2 - v1;
    float3 dir13 = v3 - v1;
    float3 dir23 = v3 - v2;

    cache.unitDir12 = normalize(dir12);
    cache.unitDir13 = normalize(dir13);
    cache.unitDir23 = normalize(dir23);

    int axisX = (axis+1)%3; // Not really the x axis
    int axisY = (axis+2)%3; // Not really the y axis

    cache.projUnitDir12 = toFloat2(cache.unitDir12, axisX, axisY);
    cache.projUnitDir13 = toFloat2(cache.unitDir13, axisX, axisY);
    cache.projUnitDir23 = toFloat2(cache.unitDir23, axisX, axisY);

    float3 slDir;
    if(get(v1, axis) != get(v3, axis))
    {
      slDir = cross(dir12, dir13);
      float z = get(slDir, axis);
      slDir = -slDir * sign(z) / length(toFloat2(slDir, axisX, axisY));
      set(slDir, axis, abs(1/get(slDir, axis)));
    }
    else
    {
      slDir = dir13 / length(toFloat2(dir13, axisX, axisY));
    }

    // exact scanline length can cause missing scanlines due to rounding error
    cache.scanlineLength = (abs(get(slDir,axisX)) + abs(get(slDir, axisY))) * 0.999;

    float2 projSlDir = toFloat2(slDir, axisX, axisY);

    // Recalculate v2 so that the scanline always start from v1
    // I.e. make dot(p2-v2, projSlDir) = 0 
    float3 v2new = v2 - cache.unitDir23 * dot(projSlDir, toFloat2(dir12, axisX, axisY)) / dot(projSlDir, cache.projUnitDir23);
    dir23 = v3 - v2new;

    cache.v1 = v1;
    cache.v2 = v2new;
    cache.v3 = v3;
    cache.slInvDotDir12 = 1 / dot(projSlDir, cache.projUnitDir12);
    cache.slInvDotDir13 = 1 / dot(projSlDir, cache.projUnitDir13);
    cache.slInvDotDir23 = 1 / dot(projSlDir, cache.projUnitDir23);
    cache.lengthDir12 = length(dir12);
    cache.lengthDir13 = length(dir13);
    cache.minLengthDir23 = length(v2 - v2new);
    cache.lengthDir23 = length(v3 - v2new);
    cache.scanlineInvDirAxis = abs(1 / get(slDir, axis));
    cache.scanlineMaxLength = dot(projSlDir, toFloat2(dir13, axisX, axisY));
  }

  __device__
  void voxelizeScanline(const ScanlineCache& cache, float slLength, int axis, float height, unsigned char color)
  {
    float3 fromPos = cache.v1;
    float3 fromDir = cache.unitDir12;
    float invDot = cache.slInvDotDir12;

    if(slLength * cache.slInvDotDir12 >= cache.lengthDir12 || slLength * cache.slInvDotDir12 < 0)
    {
      // If this also out of range assume we are outside the triangle
      if(slLength * cache.slInvDotDir23 >= cache.lengthDir23 || 
          slLength * cache.slInvDotDir23 <= cache.minLengthDir23)
      {
        return;
      }

      fromPos = cache.v2;
      fromDir = cache.unitDir23;
      invDot = cache.slInvDotDir23;
    }

    float3 from = fromPos + fromDir * slLength * invDot;
    float3 to = cache.v1 + cache.unitDir13 * slLength * cache.slInvDotDir13;

    set(from, axis, height);
    set(to, axis, height);

    voxelizeLine(from, to, color);
  }

  __device__
  void voxelizeInterior(float3 v1, float3 v2, float3 v3, int axis)
  {
    float nextPlane = floor(get(v1, axis) + 1);

    ScanlineCache cache;
    calculateScanline(v1, v2, v3, axis, cache);

    float planeT = (nextPlane - get(v1, axis)) * cache.scanlineInvDirAxis;
    float t = cache.scanlineLength;

    float plane = nextPlane-0.5;

    // Triangle forms a line
    if(std::isnan(cache.scanlineMaxLength))
    {
      return;
    }

    while(true)
    {
      t = min(t, min(planeT, cache.scanlineMaxLength));
      if(t == cache.scanlineMaxLength) // Fully voxelized
        break;
      if(t == planeT) // New scanline slice 
      {
        voxelizeScanline(cache, t, axis, plane, 1);
        plane++;
        planeT += cache.scanlineInvDirAxis;
      }
      voxelizeScanline(cache, t, axis, plane, 1);
      t += cache.scanlineLength;
    }
  }

  __device__
  void voxelize(float3 v1, float3 v2, float3 v3)
  {
    float3 normal = abs(cross(v2 - v1, v3 - v1));
    float domNormal = max(normal);
    int axis = 0;
    if(domNormal == normal.x) axis = 0;
    else if(domNormal == normal.y) axis = 1;
    else if(domNormal == normal.z) axis = 2;

    // ---- Sort by most dominant axis
    if(get(v1, axis) > get(v2, axis))
      swap(v1,v2);
    if(get(v1, axis) > get(v3, axis))
      swap(v1,v3);
    if(get(v2, axis) > get(v3, axis))
      swap(v2,v3);


    voxelizeLine(v1, v2, 1);
    voxelizeLine(v2, v3, 1);
    voxelizeLine(v1, v3, 1);

    voxelizeInterior(v1, v2, v3, axis);
  }
}
