#include "hip/hip_runtime.h"
#pragma once

#include "Common.cu"

#include "VectorMath.cu"

struct LineData
{
  int3 pos;
  int3 end;
  int3 signDir;
  int axis;
  int axisX;
  int axisY;

  int eX;
  int eY;
  int state;

  int deX;
  int deY;
  int deAxis;
  int3 nextPos;
};

__device__
int3 GetNextVoxelInLine(LineData& data)
{
  // This state based system is bad for branching prediction...
  // But I don't know how to implement getting the next voxel otherwise.
  // And the scanline algorithm requires it in the interior voxelization.

  int3 lastVoxel = data.pos;
  data.pos = data.nextPos;

  // State0
  if(data.state == 0)
  {
    data.state = 1;
    get(data.nextPos, data.axis) += get(data.signDir, data.axis);
    if(data.pos != lastVoxel)
      return data.pos;
    data.pos = data.nextPos;
  }

  // State1
  if(data.state == 1)
  {
    data.state = 2;
    int lastEX = data.eX;
    data.eX += data.deX;
    if(lastEX >= 0)
    {
      data.eX -= data.deAxis;
      get(data.nextPos, data.axisX) += get(data.signDir, data.axisX);
      if(2*lastEX > data.deX)
      {
        data.pos = lastVoxel;
        get(data.pos, data.axisX) += get(data.signDir, data.axisX);
      }
      if(data.pos != lastVoxel)
        return data.pos;
      data.pos = data.nextPos;
    }
  }
  // State2
  if(data.state == 2)
  {
    data.state = 0;
    int lastEY = data.eY;
    data.eY += data.deY;
    if(lastEY >= 0)
    {
      data.eY -= data.deAxis;
      get(data.nextPos, data.axisY) += get(data.signDir, data.axisY);
      if(2*lastEY > data.deY)
      {
        data.pos = lastVoxel;
        get(data.pos, data.axisY) += get(data.signDir, data.axisY);
      }
      if(data.pos != lastVoxel)
        return data.pos;
      data.pos = data.nextPos;
    }
  }

  // State0
  data.state = 1;
  get(data.nextPos, data.axis) += get(data.signDir, data.axis);
  return data.pos;
}

__device__
LineData CreateLineData(int3 v1, int3 v2)
{
  LineData data;
  int3 diff = v2 - v1;
  data.pos = v1;
  data.nextPos = data.pos;
  data.end = v2;
  data.signDir = sign(diff);
  diff = abs(diff);
  int maxDistance = max(diff);
  if(maxDistance == diff.x) data.axis = 0;
  else if(maxDistance == diff.y) data.axis = 1;
  else if(maxDistance == diff.z) data.axis = 2;
  data.axisX = (data.axis+1) % 3;
  data.axisY = (data.axis+2) % 3;
  if(data.axisX > data.axisY)
  {
    int tmp = data.axisX;
    data.axisX = data.axisY;
    data.axisY = tmp;
  }
  data.eX = 2 * get(diff, data.axisX) - get(diff, data.axis);
  data.eY = 2 * get(diff, data.axisY) - get(diff, data.axis);
  data.state = 0;

  data.deX = 2 * get(diff, data.axisX);
  data.deY = 2 * get(diff, data.axisY);
  data.deAxis = 2 * get(diff, data.axis);

  return data;
}

__device__
void voxelizeLine(int3 v1, int3 v2, unsigned char color)
{
  LineData data = CreateLineData(v1,v2);
  int3 lastVoxel = data.pos;

  while(data.pos != data.end)
  {
    lastVoxel = data.pos;
    voxelizePoint(data.pos, color);
    get(data.pos, data.axis) += get(data.signDir, data.axis);

    int lastEX = data.eX;
    data.eX += data.deX;
    if(lastEX >= 0)
    {
      int diff = get(data.signDir, data.axisX);

      if(2*lastEX <= data.deX)
        lastVoxel = data.pos;
      else
        get(lastVoxel, data.axisX) += diff;

      voxelizePoint(lastVoxel, color);
      get(data.pos, data.axisX) += diff;
      data.eX -= data.deAxis;
    }

    int lastEY = data.eY;
    data.eY += data.deY;
    if(lastEY >= 0)
    {
      int diff = get(data.signDir, data.axisY);

      if(2*lastEY <= data.deY)
        lastVoxel = data.pos;
      else
        get(lastVoxel, data.axisY) += diff;

      voxelizePoint(lastVoxel, color);
      get(data.pos, data.axisY) += diff;
      data.eY -= data.deAxis;
    }
  }
  voxelizePoint(data.pos, color);
}
