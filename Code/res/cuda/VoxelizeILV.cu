#include "hip/hip_runtime.h"
#pragma once

#include "Common.cu"

#include "VectorMath.cu"

struct LineData
{
  int3 t;
  int3 tStep;
  int3 signDir;
  int3 pos;
  int3 end;
};

__device__
int3& GetNextVoxelInLine(LineData& data)
{
  int tmin = min(data.t);
  int axis = 0;

  if(tmin == data.t.x && data.signDir.x != 0) axis = 0;
  else if(tmin == data.t.y && data.signDir.y != 0) axis = 1;
  else if(tmin == data.t.z && data.signDir.z != 0) axis = 2;

  data.t -= tmin;

  get(data.pos, axis) += get(data.signDir, axis);
  get(data.t, axis) = get(data.tStep, axis);
  return data.pos;
}

__device__
LineData CreateLineData(int3 v1, int3 v2)
{
  LineData data;
  int3 dir = v2 - v1;
  data.signDir = sign(dir);
  data.pos = v1;
  data.end = v2;

  int3 dir1 = make_int3(max(abs(dir.x), 1),max(abs(dir.y), 1),max(abs(dir.z), 1));

  data.t = make_int3(dir1.y * dir1.z, dir1.x * dir1.z, dir1.x * dir1.y);
  if(dir.x == 0) data.t.x = INT_MAX;
  if(dir.y == 0) data.t.y = INT_MAX;
  if(dir.z == 0) data.t.z = INT_MAX;
  data.tStep = data.t * 2;
  return data;
}

__device__
void voxelizeLine(int3 v1, int3 v2, unsigned char color)
{
  LineData data = CreateLineData(v1,v2);
  voxelizePoint(data.pos, color);

  while(data.pos != data.end)
  {
    voxelizePoint(GetNextVoxelInLine(data), color);
  }
}
