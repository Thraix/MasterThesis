#include "hip/hip_runtime.h"
#pragma once

#include "VectorMath.cu"

surface<void, 3> voxelGrid;
float* voxelGridArray;
__constant__ int voxelGridSize;
__constant__ Mat4 modelMatrix;

__shared__ bool compareFirst;

#ifdef _ERROR_ANALYSIS
__device__
void voxelizePoint(int3 v, unsigned char color)
{
  if(v.x >= 0 && v.x < voxelGridSize &&
      v.y >= 0 && v.y < voxelGridSize &&
      v.z >= 0 && v.z < voxelGridSize)
  {
    if(compareFirst)
    {
      color = 2;
    }
    else
    {
      unsigned char oldColor;
      surf3Dread(&oldColor, voxelGrid, v.x, v.y, v.z);
      if(oldColor == 3 || oldColor == 2)
        color = 3;
      else
        color = 4;
    }
    // My auto completer doesn't recognize surf3Dread or surf3Dwrite
    // So disable auto complete here
#ifndef __YCM__
    surf3Dwrite(color, voxelGrid, v.x, v.y, v.z);
#endif
  }
}
#else
__device__
void voxelizePoint(int3 v, unsigned char color)
{
  if(v.x >= 0 && v.x < voxelGridSize &&
      v.y >= 0 && v.y < voxelGridSize &&
      v.z >= 0 && v.z < voxelGridSize)
  {
    // My auto completer doesn't recognize surf3Dread or surf3Dwrite
    // So disable auto complete here
#ifndef __YCM__
    /* color = 1; */
    /* unsigned char oldColor; */
    /* surf3Dread(&oldColor, voxelGrid, v.x, v.y, v.z); */
    /* oldColor++; */
    /* color = oldColor; */
    /* if(oldColor == 0) */
    surf3Dwrite(color, voxelGrid, v.x, v.y, v.z);
#endif
  }
}
#endif
