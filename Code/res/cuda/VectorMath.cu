#pragma once
#include <hip/hip_runtime.h>


struct Mat4
{
  float elements[16];

  __device__ float operator[](int i)
  {
    return elements[i];
  }
};

__device__ float3 operator+(float3 v1, float3 v2)
{
  return make_float3(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
}

__device__ float3 operator+(float3 v, float f)
{
  return make_float3(v.x + f, v.y + f, v.z + f);
}

__device__ float3 operator-(float3 v1, float3 v2)
{
  return make_float3(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
}

__device__ float3 operator*(float3 v1, float3 v2)
{
  return make_float3(v1.x * v2.x, v1.y * v2.y, v1.z * v2.z);
}

__device__ float3 operator*(float3 v, float f)
{
  return make_float3(v.x * f, v.y * f, v.z * f);
}
__device__ float3 operator-(float3 v)
{
  return make_float3(-v.x, -v.y, -v.z);
}

__device__ float3 operator/(float3 v1, float3 v2)
{
  return make_float3(v1.x / v2.x, v1.y / v2.y, v1.z / v2.z);
}

__device__ float3 operator/(float3 v, float f)
{
  return make_float3(v.x / f, v.y / f, v.z / f);
}

__device__ float3 operator/(float f, float3 v)
{
  return make_float3(f / v.x, f / v.y, f / v.z);
}
__device__ bool operator!=(float3 v1, float3 v2)
{
  return v1.x != v2.x || v1.y != v2.y || v1.z != v2.z;
}

__device__ bool operator!=(int3 v1, int3 v2)
{
  return v1.x != v2.x || v1.y != v2.y || v1.z != v2.z;
}

__device__ bool operator==(float3 v1, float3 v2)
{
  return v1.x == v2.x && v1.y == v2.y && v1.z == v2.z;
}

__device__ bool operator==(int3 v1, int3 v2)
{
  return v1.x == v2.x && v1.y == v2.y && v1.z == v2.z;
}

__device__ void operator-=(float3& v, float f)
{
  v.x -= f;
  v.y -= f;
  v.z -= f;
}

__device__ float3 operator*(Mat4& m, float3 v)
{
  float x = m[0] * v.x + m[4] * v.y + m[8]  * v.z + m[12];
  float y = m[1] * v.x + m[5] * v.y + m[9]  * v.z + m[13];
  float z = m[2] * v.x + m[6] * v.y + m[10] * v.z + m[14];
  float w = m[3] * v.x + m[7] * v.y + m[11] * v.z + m[15];
  return make_float3(x, y, z);
}

__device__ float get(float3 v, int index)
{
  float* ptr = (float*)&v;
  return *(ptr + index);
}

__device__ void set(float3& v, int index, float value)
{
  float* ptr = (float*)&v;
  *(ptr + index) = value;
}

__device__ float sign(float f)
{
  return f < 0.0f ? -1.0f : (f > 0.0f ? 1.0f : 0.0f);
}

__device__ float3 sign(float3 v)
{
  return make_float3(sign(v.x), sign(v.y), sign(v.z));
}

__device__ float min(float3 v)
{
  return min(v.x, min(v.y, v.z));
}

__device__ float max(float3 v)
{
  return max(v.x, max(v.y, v.z));
}

__device__ float3 abs(float3 v)
{
  return make_float3(abs(v.x), abs(v.y), abs(v.z));
}

__device__ int3 toInt3(float3 v)
{
  return make_int3(floor(v.x), floor(v.y), floor(v.z));
}

__device__ float3 toFloat3(int3 v)
{
  return make_float3(float(v.x), float(v.y), float(v.z));
}

__device__ float3 cross(float3 v1, float3 v2)
{
  return make_float3(
      v1.y * v2.z - v1.z * v2.y,
      v1.z * v2.x - v1.x * v2.z,
      v1.x * v2.y - v1.y * v2.x);
}

__device__ float dot(float3 v1, float3 v2)
{
  return v1.x * v2.x + v1.y * v2.y + v1.z * v2.z;
}

__device__ float length(float3 v)
{
  return sqrt(v.x * v.x + v.y * v.y + v.z * v.z);
}

__device__ float3 normalize(float3 v)
{
  return v / length(v);
}

__device__ void swap(float3& v1, float3& v2)
{
  float3 tmp = v1;
  v1 = v2;
  v2 = tmp;
}

__device__
int2 operator*(int2 v1, int2 v2)
{
  return make_int2(v1.x * v2.x, v1.y * v2.y);
}

__device__
int3 operator+(int3 v1, int3 v2)
{
  return make_int3(v1.x + v2.x, v1.y + v2.y, v1.z + v2.z);
}

__device__
int3 operator-(int3 v1, int3 v2)
{
  return make_int3(v1.x - v2.x, v1.y - v2.y, v1.z - v2.z);
}

__device__
int3 operator*(int3 v1, int3 v2)
{
  return make_int3(v1.x * v2.x, v1.y * v2.y, v1.z * v2.z);
}

__device__
int3 operator*(int3 v, int i)
{
  return make_int3(v.x * i, v.y * i, v.z * i);
}

__device__
void operator-=(int3& v, int i)
{
  v.x -= i;
  v.y -= i;
  v.z -= i;
}

__device__ int& get(int3& v, int index)
{
  return *((int*)&v + index);
}

__device__ void set(int3& v, int index, int value)
{
  int* ptr = ((int*)&v + index);
  *ptr = value;
}


__device__ 
int2 toInt2(int3 v, int axis1, int axis2)
{
  return make_int2(get(v, axis1), get(v, axis2));
}

__device__
int min(int3 v)
{
  return min(v.x, min(v.y, v.z));
}

__device__
int sign(int i)
{
  return i < 0 ? -1 : (i > 0 ? 1 : 0);
}

__device__
int3 sign(int3 v)
{
  return make_int3(sign(v.x), sign(v.y), sign(v.z));
}

__device__
int3 abs(int3 v)
{
  return make_int3(abs(v.x), abs(v.y), abs(v.z));
}

__device__
int dot(int2 v1, int2 v2)
{
  return v1.x * v2.x + v1.y * v2.y;
}

__device__
int3 cross(int3 v1, int3 v2)
{
  return make_int3(
      v1.y * v2.z - v1.z * v2.y,
      v1.z * v2.x - v1.x * v2.z,
      v1.x * v2.y - v1.y * v2.x);
}

__device__ int max(int3 v)
{
  return max(v.x, max(v.y, v.z));
}

__device__
int sum(int2 v)
{
  return v.x + v.y;
}

__device__ 
void swap(int3& v1, int3& v2)
{
  int3 tmp = v1;
  v1 = v2;
  v2 = tmp;
}

__device__ float2 operator+(float2 v1, float2 v2)
{
  return make_float2(v1.x + v2.x, v1.y + v2.y);
}

__device__ float2 operator-(float2 v1, float2 v2)
{
  return make_float2(v1.x - v2.x, v1.y - v2.y);
}

__device__ float2 operator*(float2 v, float f)
{
  return make_float2(v.x * f, v.y * f);
}

__device__ float2 operator/(float2 v, float f)
{
  return make_float2(v.x / f, v.y / f);
}

__device__ float& get(float2& v, int index)
{
  float* ptr = (float*)&v;
  return *(ptr + index);
}

__device__ float length(float2 v)
{
  return sqrt(v.x * v.x + v.y * v.y);
}

__device__ float2 toFloat2(float3 v, int axis1, int axis2)
{
  return make_float2(get(v, axis1), get(v, axis2));
}

__device__ float2 normalize(float2 v)
{
  return v / length(v);
}

__device__ float dot(float2 v1, float2 v2)
{
  return v1.x * v2.x + v1.y * v2.y;
}
